#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCKSIZE 1024
#define MAXIT 1
#define TOTROWS		(BLOCKSIZE*8)
#define TOTCOLS		(BLOCKSIZE*8)
#define NOTSETLOC       -1 // for cells that are not fixed

#define QMAX(x,y) (((x) > (y))? (x): (y))


int *lkeepgoing;
float *iplate;
float *oplate;
float *fixed;
float *tmp;
int ncols, nrows;

double When();
void Compute();


int main(int argc, char *argv[])
{
	double t0, tottime;
	ncols = TOTCOLS;
	nrows = TOTROWS;

	hipMalloc((void **) &lkeepgoing, nrows * ncols * sizeof(int));
	hipMalloc((void **) &iplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &oplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &fixed,  nrows * ncols * sizeof(float));
	fprintf(stderr,"Memory allocated\n");

	t0 = When();
	/* Now proceed with the Jacobi algorithm */
	Compute();

	tottime = When() - t0;
	printf("Total Time is: %lf sec.\n", tottime);

	return 0;
}

__global__ void InitArrays(float *ip, float *op, float *fp, int *kp, int ncols)
{
	int i;
	float *fppos, *oppos, *ippos;
        int *kppos;
        int blockOffset;
        int rowStartPos;
        int colsPerThread;
	
        // Each block gets a row, each thread will fill part of a row

	// Calculate the offset of the row
        blockOffset = blockIdx.x * ncols;
        // Calculate our offset into the row
	rowStartPos = threadIdx.x * (ncols/blockDim.x);
        // The number of cols per thread
        colsPerThread = ncols/blockDim.x;

	ippos = ip + blockOffset+ rowStartPos;
	fppos = fp + blockOffset+ rowStartPos;
	oppos = op + blockOffset+ rowStartPos;
	kppos = kp + blockOffset+ rowStartPos;

	for (i = 0; i < colsPerThread; i++) {
		fppos[i] = NOTSETLOC; // Not Fixed
		ippos[i] = 50;
		oppos[i] = 50;
	        kppos[i] = 1; // Keep Going
	}
        // Insert code to set the rest of the boundary and fixed positions
}
__global__ void doCalc(float *iplate, float *oplate, int ncols)
{
	/* Compute the 5 point stencil for my region */
}

__global__ void doCheck(float *iplate, float *oplate, float *fixed, int *lkeepgoing, int ncols)
{
	// Calculate keepgoing array
}

__global__ void reduceSingle(int *idata, int *single, int nrows)
{
	// Reduce rows to the first element in each row
	int i;
    int rowStartPos;
    int colsPerThread;
    int *mypart;
    extern __shared__ int parts[];
	
    // Each block gets a row, each thread will reduce part of a row

        // Calculate our offset into the row
	rowStartPos = threadIdx.x * (nrows/blockDim.x);
        // The number of cols per thread
    colsPerThread = nrows/blockDim.x;

	//mypart = idata + blockOffset + rowStartPos;

	// sum my part of one dim array
	parts[threadIdx.x] = 0;
	for (i = rowStartPos; i < colsPerThread + start; i++) {
		parts[threadIdx.x] += idata[i];
	}
	int tid = threadIdx.x
	if(tid <512) { parts[tid] += parts[tid+512];}
	__syncthreads();
	if(tid <256) { parts[tid] += parts[tid+256];}
	__syncthreads();
	if(tid <128) { parts[tid] += parts[tid+128];}
	__syncthreads();
	if(tid <64) { parts[tid] += parts[tid+64];}
	__syncthreads();
	if(tid <32) { parts[tid] += parts[tid+32];}
	__syncthreads();
	if(tid == 0) {
		for(i=0;i<32;i++) {
			*single += parts[i];
		}
	}
}
__global__ void reduceSum(int *idata, int *odata, unsigned int ncols)
{
	// Reduce rows to the first element in each row
	int i;
    int blockOffset;
    int rowStartPos;
    int colsPerThread;
    int *mypart;
	
    // Each block gets a row, each thread will reduce part of a row

	// Calculate the offset of the row
        blockOffset = blockIdx.x * ncols;
        // Calculate our offset into the row
	rowStartPos = threadIdx.x * (ncols/blockDim.x);
        // The number of cols per thread
        colsPerThread = ncols/blockDim.x;

	mypart = idata + blockOffset + rowStartPos;

	// Sum all of the elements in my thread block and put them 
        // into the first column spot
	for (i = 1; i < colsPerThread; i++) {
		mypart[0] += mypart[i];
	}
	__syncthreads(); // Wait for everyone to complete
        // Now reduce all of the threads in my block into the first spot for my row
	if(threadIdx.x == 0) {
		odata[blockIdx.x] = 0;
		for(i = 0; i < blockDim.x; i++) {
			odata[blockIdx.x] += mypart[i*colsPerThread];
		}
	}
	// We cant synchronize between blocks, so we will have to start another kernel
}
	
void Compute()
{
	int *keepgoing_single;
	int *keepgoing_sums;
	int keepgoing;
	int blocksize = BLOCKSIZE;
	int iteration;

	ncols = TOTCOLS;
	nrows = TOTROWS;

	// One block per row
	InitArrays<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols);
	hipMalloc((void **)&keepgoing_single, 1 * sizeof(int));
	keepgoing = 1;
	hipMalloc((void **)&keepgoing_sums, nrows * sizeof(int));
 	int *peek = (int *)malloc(nrows*sizeof(int));

	for (iteration = 0; (iteration < MAXIT) && keepgoing; iteration++)
	{
		doCalc<<< nrows, blocksize >>>(iplate, oplate, ncols);
		doCheck<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols);
		//reduce value to first cell in each row`
        	reduceSum<<< nrows, blocksize>>>(lkeepgoing, keepgoing_sums, ncols);
		// hipMemcpy(peek, keepgoing_sums, nrows*sizeof(int), hipMemcpyDeviceToHost);
		// fprintf(stderr, "after hipMemcpy \n");
		// int i;
 	// 	for(i = 0; i < nrows; i++) {
		// 	fprintf(stderr, "%d, ",peek[i]);
		// }
		// Now we have the sum for each row in the first column, 
		//  reduce to one value
		double t0 = When();
		int singleLoop;
		for(singleLoop = 0; singleLoop < 1000; singleLoop++) {}
			reduceSingle<<<1, blocksize, blocksize*sizeof(int)>>>(keepgoing_sums, keepgoing_single);
		}
		printf("reduce single:%f\n", When() - t0);
		keepgoing = 0;
		hipMemcpy(&keepgoing, keepgoing_single, 1 * sizeof(int), hipMemcpyDeviceToHost);
		fprintf(stderr, "keepgoing = %d\n", keepgoing);

		/* swap the new value pointer with the old value pointer */
		tmp = oplate;
		oplate = iplate;
		iplate = tmp;
	}
	free(peek);
	hipFree(keepgoing_single);
	hipFree(keepgoing_sums);
	fprintf(stderr,"Finished in %d iterations\n", iteration);
}

/* Return the current time in seconds, using a double precision number.       */
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}
